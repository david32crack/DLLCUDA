#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <windows.h>
#include "kernel.h"


byte* dev_src2 = NULL;
byte* dev_dst2 = NULL;
byte* dev_aux2 = NULL;

int threadsInX;
int threadsInY;
int blocksInX;
int blocksInY;

int width;
int height;
int size_img;



//**************************************************** Funciones GPU **********************************************************//

__global__ void threshold(byte* src, byte* dst, byte min, byte max, int stride, int size)
{
	for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += stride)
		dst[pos] = (src[pos] >= min && src[pos] <= max) ? 1 : 0;
}


__global__ void erode(byte* src, byte* dst, int w, int h, int radio)
{
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx >= w || posy >= h)
		return;

	unsigned int start_y = max(posy - radio, 0);
	unsigned int end_y = min(h - 1, posy + radio);
	unsigned int start_x = max(posx - radio, 0);
	unsigned int end_x = min(w - 1, posx + radio);

	int _min = 255;

	for (int y = start_y; y <= end_y; y++)
		for (int x = start_x; x <= end_x; x++)
			_min = min(_min, src[y * w + x]);

	dst[posy * w + posx] = _min;
}

__global__ void erode_separable_step2(byte* src, byte* dst, int w, int h, int radio) {

	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx >= w || posy >= h)
		return;

	unsigned int start_y = max(posy - radio, 0);
	unsigned int end_y = min(h - 1, posy + radio);

	int _min = 255;
	for (int y = start_y; y <= end_y; y++) {
		_min = min(_min, src[y * w + posx]);
	}
	dst[posy * w + posx] = _min;
}

__global__ void erode_separable_step1(byte* src, byte* dst, int w, int h, int radio) {

	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx >= w || posy >= h)
		return;

	unsigned int start_x = max(posx - radio, 0);
	unsigned int end_x = min(w - 1, posx + radio);

	int _min = 255;
	for (int x = start_x; x <= end_x; x++) {
		_min = min(_min, src[posy * w + x]);
	}
	dst[posy * w + posx] = _min;

}

__global__ void dilate(byte * src, byte *dst, int w, int h, int radio)
{
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx >= w || posy >= h)
		return;

	unsigned int start_y = max(posy - radio, 0);
	unsigned int end_y = min(h - 1, posy + radio);
	unsigned int start_x = max(posx - radio, 0);
	unsigned int end_x = min(w - 1, posx + radio);

	int _max = 0;

	for (int y = start_y; y <= end_y; y++)
		for (int x = start_x; x <= end_x; x++)
			_max = max(_max, src[y * w + x]);

	dst[posy * w + posx] = _max;
}

__global__ void dilate_separable_step2(byte* src, byte* dst, int w, int h, int radio) {

	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx >= w || posy >= h)
		return;

	unsigned int start_y = max(posy - radio, 0);
	unsigned int end_y = min(h - 1, posy + radio);

	int _max = 0;
	for (int y = start_y; y <= end_y; y++) {
		_max = max(_max, src[y * w + posx]);
	}
	dst[posy * w + posx] = _max;
}

__global__ void dilate_separable_step1(byte* src, byte* dst, int w, int h, int radio) {

	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx >= w || posy >= h)
		return;

	unsigned int start_x = max(posx - radio, 0);
	unsigned int end_x = min(w - 1, posx + radio);

	int _max = 0;
	for (int x = start_x; x <= end_x; x++) {
		_max = max(_max, src[posy * w + x]);
	}
	dst[posy * w + posx] = _max;

}

__global__ void reverseThreshold(byte* src, byte* dst, byte min, byte max, int stride, int size)
{
	for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += stride)
		dst[pos] = (src[pos] >= min && src[pos] <= max) ? 0 : 1;
}





//********************************************************* Llamadas a GPU *************************************************************//

void dev_threshold(byte *src, byte *dst, byte min, byte max, int threads, int blocks, int stride, int size, int* error) {

	hipError_t cudaStatus;

	threshold << < blocks, threads >> > (src, dst, min, max, stride, size);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		*error = -7;
}

void dev_erode(byte *src, byte *dst, int width, int height, int radio, dim3 threads, dim3 blocks, int* error) {

	hipError_t cudaStatus;

	erode << < blocks, threads >> > (src, dst, width, height, radio);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		*error = -8;

}

void dev_erode_twoSteps(byte* src, byte* dst, byte* aux, int radio, dim3 threads, dim3 blocks, int* error) {

	hipError_t cudaStatus;

	erode_separable_step1 << <blocks, threads >> > (src, aux, width, height, radio);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		*error = -10;
		return;
	}
		
	erode_separable_step2 << <blocks, threads >> > (aux, dst, width, height, radio);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
		*error = -10;

}

void dev_dilate(byte *src, byte *dst, int width, int height, int radio, dim3 threads, dim3 blocks, int* error) {

	hipError_t cudaStatus;

	dilate << < blocks, threads >> > (src, dst, width, height, radio);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		*error = -11;
}

void dev_dilate_twoSteps(byte* src, byte* dst, byte* aux, int radio, dim3 threads, dim3 blocks, int* error) {

	hipError_t cudaStatus;

	dilate_separable_step1 << <blocks, threads >> > (src, aux, width, height, radio);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		*error = -14;
		return;
	}

	dilate_separable_step2 << <blocks, threads >> > (aux, dst, width, height, radio);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		*error = -14;

}

void dev_reverseThreshold(byte* src, byte* dst, byte min, byte max, int threads, int blocks, int stride, int size, int* error) {

	hipError_t cudaStatus;

	reverseThreshold << < blocks, threads >> > (src, dst, min, max, stride, size);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		*error = -12;
}




//******************************************************* Herramientas para GPU ********************************************************//


// DETERMINA EL NUMERO DE DEVICES DISPONIBLES
void expert_numAvailableDevices(int *numCUDADevices)
{
	hipError_t cudaStatus = hipGetDeviceCount(numCUDADevices);
	if (cudaStatus != hipSuccess)
		*numCUDADevices = 0;
}

// RESETEA EL DEVICE SELECCIONADO
void  expert_resetDevice(int deviceId, int* error)
{
	int numDevices;
	expert_numAvailableDevices(&numDevices);

	if (deviceId < 0 || deviceId >= numDevices)
		*error = -222;
	else
	{
		hipError_t cudaStatus = hipSetDevice(deviceId);

		if (cudaStatus == hipSuccess)
			cudaStatus = hipDeviceReset();

		if (cudaStatus == hipSuccess)
			*error = 0;
		else
			*error = -333;
	}
}

// ESTABLECE EL DEVICE SELECCIONADO 
void  expert_setDevice(int deviceId, int *error)
{
	int numDevices;
	expert_numAvailableDevices(&numDevices);
	if (deviceId < 0 || deviceId >= numDevices)
		*error = -222;
	else
	{
		hipError_t cudaStatus = hipSetDevice(deviceId);
		if (cudaStatus == hipSuccess)
			*error = 0;
		else
			*error = -444;
	}
}

// RESETEA TODOS LOS DEVICES DISPONIBLES
void  expert_resetAllDevices(int* error)
{
	int numDevices;
	expert_numAvailableDevices(&numDevices);

	*error = 0;
	if (numDevices < 1)
		*error = -111;
	else
	{
		for (int dev = 0; dev < numDevices && *error == 0; dev++)
			expert_resetDevice(dev, error);
		expert_setDevice(0, error);
	}
}



/*string expert_descriptionError(int* error) {

	string error_msg;
	switch (*error) {
	case -111:
		error_msg = "Al intentar resetear todos los devices, no hab�a devices";
		break;
	case -222:
		error_msg = "Numero del device metido no esta en el rango";
		break;
	case -333:
		error_msg = "Error en reseteo del device metido";
		break;
	case -444:
		error_msg = "Error en el establecimiento del device";
		break;
	case -1:
		error_msg = "Al intentar reservar memoria para la imagen fuente";
		break;
	case -2:
		error_msg = "Al intentar reservar memoria para la imagen destino";
		break;
	case -3:
		error_msg = "Copia de memoria de CPU a GPU";
		break;
	case -4:
		error_msg = "Copia de memoria de GPU a CPU";
		break;
	case -5:
		error_msg = "Al intentar liberar memoria para la imagen fuente";
		break;
	case -6:
		error_msg = "Al intentar liberar memoria para la imagen destino";
		break;
	case -7:
		error_msg = "Falla en threshold";
		break;
	case -8:
		error_msg = "Falla en Erode Low";
		break;
	case -9:
		error_msg = "Falla en Erode Fast(TwoSteps) al intentar reservar memoria en GPU";
		break;
	case -10:
		error_msg = "Falla en Erode Fast(TwoSteps) al hacer el algoritmo";
		break;
	case -11:
		error_msg = "Falla en Dilate";
		break;
	case -12:
		error_msg = "Falla en ReverseThreshold";
		break;
	case -13:
		error_msg = "Falla en Dilate Fast(TwoSteps) al intentar reservar memoria en GPU";
		break;
	case -14:
		error_msg = "Falla en Dilate Fast(TwoSteps) al hacer algoritmo";
		break;
	case -15:
		error_msg = "Fallo en reescaldo de imagen";
		break;
	default: 
		error_msg = "NO ERROR";
	}
	return error_msg;
}
*/



//************************************************ Determinar Threads Y Bloques *********************************************************//

void setNumberThreads1D(int threadsX, int blocksX, bool automatic) {
	
	if (automatic) {

		threadsInX = 1024;
		blocksInX = 640;  ///////// 65535 / 1024 = 64 * 10

	}
	else {

		if (threadsX == 0 || threadsX > 1024)
			threadsX = 1024;
		if (blocksX == 0 || blocksX > 65535)
			blocksX = 65535;

		threadsInX = threadsX;
		blocksInX = blocksX;
	}

}

void setNumberThreads2D(int threadsX, int threadsY, int blocksX, int blocksY, bool automatic) {

	if (automatic) {

		if (width > 3000) {
			threadsInX = 16;
			blocksInX = 500;
		}
		else if (width > 1600) {
			threadsInX = 8;
			blocksInX = 400;
		}
		else {
			threadsInX = 8;
			blocksInX = 240;
		}

		if (height > 3000){
			threadsInY = 16;
			blocksInY = 500;
		}

		else if (width > 1600) {
			threadsInY = 8;
			blocksInY = 400;
		}

		else {
			threadsInY = 8;
			blocksInY = 240;
		}

	}
	else {

		if (threadsX == 0 || threadsX > 512)
			threadsX = 512;
		if (blocksX == 0 || blocksX > 65535)
			blocksX = 65535;

		threadsInX = threadsInY = threadsX;
		blocksInX = blocksInY = blocksX;

	}
	
	//dim3 grid(width / threadsPerBlock.x, height / threadsPerBlock.y);
}

void setDimensionNumber_Threads_Blocks(int size, int threadsX, int threadsY, int blocksX, int blocksY, bool automatic) {

	if (size > 2 || size <= 0)
		size = 1;
	if (size == 1)
		setNumberThreads1D(threadsX, blocksX, automatic);
		
	else
		setNumberThreads2D(threadsX, threadsY, blocksX, blocksY, automatic);
}




//***************************************************** INTERCAMBIAR ARRAYS *********************************************************//

void swapBuffers(byte** a, byte** b)
{
	byte* aux = *a;
	*a = *b;
	*b = aux;
}


//***********************************************************************************************************************************//
/* ERRORES de procesamiento:
/* 
 *
 * -111: Al intentar resetear todos los devices, no hab�a devices
 * -222: Numero del device metido no esta en el rango
 * -333: Error en reseteo del device metido
 * -444: Error en el establecimiento del device
 *
 *   -1: Al intentar reservar memoria para la imagen fuente
 *   -2: Al intentar reservar memoria para la imagen destino
 *   -3: Copia de memoria de CPU a GPU
 *   -4: Copia de memoria de GPU a CPU
 *   -5: Al intentar liberar memoria para la imagen fuente 
 *   -6: Al intentar liberar memoria para la imagen destino
 *   -7: Falla en threshold
 *   -8: Falla en Erode Low 
 *   -9: Falla en Erode Fast (TwoSteps) al intentar reservar memoria en GPU 
 *  -10: Falla en Erode Fast (TwoSteps) al hacer el algoritmo
 *  -11: Falla en Dilate
 *  -12: Falla en ReverseThreshold
 *  -13: Falla en Dilate Fast (TwoSteps) al intentar reservar memoria en GPU
 *  -14: Falla en Dilate Fast (TwoSteps) al hacer algoritmo
 *  -15: Fallo en reescaldo de imagen
 *  -16: Fallo en Close
 *  -17: Fallo en Open
 *
 */


//************************************ Reserva de memoria en GPU y copia de imagen en GPU ****************************************//

void reservationMemory_CopyHostToDeviceOnce(byte* src, int w, int h, int* error) {

	hipError_t cudaStatus;

	width = w;
	height = h;
	size_img = width * height;

	hipDeviceSynchronize();

	//if (dev_src2 == NULL)
	//{
	cudaStatus = hipMalloc(&dev_src2, size_img);
	if (cudaStatus != hipSuccess) {
		//printf("Error en reserva de memoria del dev_src");
		*error = -1;
		//dev_src2 = NULL;
		//dev_dst2 = NULL;
		return;
	}

	cudaStatus = hipMalloc(&dev_dst2, size_img);
	if (cudaStatus != hipSuccess) {
		//printf("Error en reserva de memoria del dev_dst");
		*error = -2;
		//dev_src2 = NULL;
		//dev_dst2 = NULL;
		hipFree(dev_src2);
		return;
	}
	//}

	cudaStatus = hipMemcpy(dev_src2, src, size_img, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		//printf("Error en copia de CPU a GPU");
		*error = -3;

		//dev_src2 = NULL;
		hipFree(dev_src2);

		//dev_dst2 = NULL;
		hipFree(dev_dst2);
	}

}
/*void reservationMemory_CopyHostToDeviceMulti(byte* src, byte* dev_src,  byte* dev_dst, int size, int* error) {
	
	hipError_t cudaStatus;

	*error = 1000;

	if (dev_src == NULL) 
	{
		cudaStatus = hipMalloc(&dev_src, size);
		if (cudaStatus != hipSuccess) {
			//printf("Error en reserva de memoria del dev_src");
			*error = -1;
			dev_src = NULL;
			dev_dst = NULL;
			return;
		}

		cudaStatus = hipMalloc(&dev_dst, size);
		if (cudaStatus != hipSuccess) {
			//printf("Error en reserva de memoria del dev_dst");
			*error = -2;
			dev_src = NULL;
			dev_dst = NULL;
			hipFree(dev_src);
		}
	}

	cudaStatus = hipMemcpy(dev_src, src, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		//printf("Error en copia de CPU a GPU");
		*error = -3;

		dev_src = NULL;
		hipFree(dev_src);

		dev_dst = NULL;
		hipFree(dev_dst);
	}		

}*/

//***********************************************************************************************************************************//




//************************************ Liberaci�n de memoria en GPU y copia de imagen en CPU ****************************************//

void freeMemory_CopyDeviceToHostOnce(byte* dst, int* error) {

	hipError_t cudaStatus;

	*error = 0;

	hipDeviceSynchronize();

	cudaStatus = hipMemcpy(dst, dev_src2, size_img, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		*error = -4;
		//dev_src2 = NULL;
		hipFree(dev_src2);

		//dev_dst2 = NULL;
		hipFree(dev_dst2);
		return;
	}

	//dev_src2 = NULL;
	hipFree(dev_src2);

	//dev_dst2 = NULL;
	hipFree(dev_dst2);

}
/*void freeMemory_CopyDeviceToHostMulti(byte* dst, byte* dev_src, byte* dev_dst, int size, int* error) {

	hipError_t cudaStatus;

	*error = 1000;

	cudaStatus = hipMemcpy(dst, dev_src, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		*error = -4;
		dev_src = NULL;
		hipFree(dev_src);

		dev_dst = NULL;
		hipFree(dev_dst);
	}

	dev_src = NULL;
	hipFree(dev_src);

	dev_dst = NULL;
	hipFree(dev_dst);

}*/

//***********************************************************************************************************************************//




//************************************************ Reserva de memoria en GPU ********************************************************//

void reservationMemoryOnce(int w, int h, int* error) {

	hipError_t cudaStatus;

	width = w;
	height = h;
	size_img = width * height;

	//if (dev_src2 == NULL)
	//{
	cudaStatus = hipMalloc(&dev_src2, size_img);
	if (cudaStatus != hipSuccess)
	{
		//printf("Error en reserva de memoria");
		*error = -1;
		//dev_src2 = NULL;
		hipFree(dev_src2);
		//dev_dst2 = NULL;
		return;
	}

	cudaStatus = hipMalloc(&dev_dst2, size_img);
	if (cudaStatus != hipSuccess)
	{
		//printf("Error en reserva de memoria");
		*error = -2;
		//dev_src2 = NULL;
		hipFree(dev_src2);
		//dev_dst2 = NULL;
		hipFree(dev_dst2);
	}	
	//}

	hipDeviceSynchronize();

}
/*void reservationMemoryMulti(byte* dev_src, byte* dev_dst, int size, string* error) {

	hipError_t cudaStatus;

	if (dev_src == NULL)
	{
		cudaStatus = hipMalloc(&dev_src, size);
		if (cudaStatus != hipSuccess) 
		{
			dev_src = NULL;
			dev_dst = NULL;
			hipFree(dev_src);
			goto Error;
		}
		
		cudaStatus = hipMalloc(&dev_dst, size);
		if (cudaStatus != hipSuccess) 
		{
			dev_src = NULL;
			dev_dst = NULL;
			hipFree(dev_src);
			hipFree(dev_dst);
			goto Error;
		}
		*error = hipGetErrorName(cudaStatus);
	}

Error:

	//printf("Error en reserva de memoria");
	*error = hipGetErrorName(cudaStatus);
}*/

//***********************************************************************************************************************************//




//********************************************* Liberacion de memoria en la GPU *****************************************************//

void freeMemoryOnce(int* error) {

	hipError_t cudaStatus;

	*error = 0;

	

	//dev_src2 = NULL;
	cudaStatus = hipFree(dev_src2);
	if (cudaStatus != hipSuccess)
		*error = -5;
	else {
		*error = 10;
	}

	//dev_dst2 = NULL;
	cudaStatus = hipFree(dev_dst2);
	if (cudaStatus != hipSuccess)
		*error = -6;
	else {
		*error = 11;
	}

	hipDeviceSynchronize();

}
/*void freeMemoryMulti(byte* dev_src, byte* dev_dst, string* error) {

	hipError_t cudaStatus;

	dev_src = NULL;
	dev_dst = NULL;

	cudaStatus = hipFree(dev_src);
	if (cudaStatus != hipSuccess)
		goto Error;

	cudaStatus = hipFree(dev_dst);
	if (cudaStatus != hipSuccess)
		goto Error;

	*error = hipGetErrorName(cudaStatus);

Error:
	*error = hipGetErrorName(cudaStatus);
}*/

//***********************************************************************************************************************************//




//******************************************** Copia de memoria de CPU a GPU ********************************************************//

void copyHostToDeviceOnce(byte* src, int* error) {

	hipError_t cudaStatus;

	hipDeviceSynchronize();

	cudaStatus = hipMemcpy(dev_src2, src, size_img, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		//printf("Error en copia de CPU a GPU");
		*error = -3;

		//dev_src2 = NULL;
		hipFree(dev_src2);

		//dev_dst2 = NULL;
		hipFree(dev_dst2);

	}

	hipDeviceSynchronize();
}
/*void copyHostToDeviceMulti(byte* src, byte* dev_src, byte* dev_dst, int size, string* error) {

	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(dev_src, src, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		goto Error;

	*error = hipGetErrorName(cudaStatus);

Error:

	//printf("Error en copia de host a device");
	*error = hipGetErrorName(cudaStatus);

	dev_src = NULL;
	dev_dst = NULL;

	hipFree(dev_src);
	hipFree(dev_dst);

}*/

//***********************************************************************************************************************************//




//********************************************** Copia de memoria de GPU a CPU ******************************************************//

void copyDeviceToHostOnce(byte *dst, int* error) {

	hipError_t cudaStatus;

	*error = 0;

	hipDeviceSynchronize();

	cudaStatus = hipMemcpy(dst, dev_src2, size_img, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		//printf("Error en copia de device a host ");
		*error = -4;

		//dev_src2 = NULL;
		hipFree(dev_src2);

		//dev_dst2 = NULL;
		hipFree(dev_dst2);
	}

	hipDeviceSynchronize();
}
/*void copyDeviceToHostMulti(byte *dst, byte* dev_src, byte* dev_dst, int size, string* error) {

	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(dst, dev_src, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		goto Error;

	*error = hipGetErrorName(cudaStatus);

Error:

	//printf("Error en copia de device a host ");
	*error = hipGetErrorName(cudaStatus);

	dev_src = NULL;
	dev_dst = NULL;

	hipFree(dev_src);
	hipFree(dev_dst);

}*/


// ************************************************* LLAMADAS A THRESHOLD ***********************************************************//


/////////////////////////////////////////// Le indicas los threads que quieres /////////////////////////////////////////

void dev_threshold_manualOnce(byte min, byte max, int threads, int blocks, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(1, threads, 0, blocks, 0, false);
	int stride = threadsInX * blocksInX;

	dev_threshold(dev_src2, dev_dst2, min, max, threadsInX, blocksInX, stride, size_img, error);
	swapBuffers(&dev_src2, &dev_dst2);
}

/*void dev_threshold_manualMulti(byte* dev_src, byte* dev_dst, int size, byte min, byte max, int threads, int blocks) {

	int stride = threads * blocks;

	if (threads == 0 || threads > 1024)
		threads = 1024;
	if (blocks == 0 || blocks > 65535)
		blocks = 65535;

	dev_threshold(dev_src, dev_dst, min, max, threads, blocks, stride, size);
	swapBuffers(&dev_src, &dev_dst);
}*/

/////////////////////////////////////////// Los threads se calculan de forma automatica /////////////////////////////////////////

void dev_threshold_automaticoOnce(byte min, byte max, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(1, 0, 0, 0, 0, true);
	int stride = threadsInX * blocksInX;

	dev_threshold(dev_src2, dev_dst2, min, max, threadsInX, blocksInX, stride, size_img, error);
	swapBuffers(&dev_src2, &dev_dst2);

}

/*void dev_threshold_automaticoMulti(byte* dev_src, byte* dev_dst, int size, byte min, byte max, int* error) {

	int threads = 1024;
	int blocks = 640;  ///////// 65535 / 1024 = 64 * 10 
	int stride = threads * blocks;

	dev_threshold(dev_src, dev_dst, min, max, threads, blocks, stride, size);
	swapBuffers(&dev_src, &dev_dst);
}*/



//****************************************************** LLAMADAS A ERODE ************************************************************//


// ERODE - Optimo MANUAL 
void dev_erode_manualOnce(int radio, int threads, int blocks, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, threads, threads, blocks, blocks, false);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	dev_erode(dev_src2, dev_dst2, width, height, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

}
/*void dev_erode_manualMulti(byte* dev_src, byte* dev_dst, int width, int height, int radio, int threads, int blocks) {

	if (threads == 0 || threads > 1024 )
		threads = 1024;
	if (blocks == 0 || blocks > 65535)
		blocks = 65535;

	dim3 threadsPerBlock(threads, threads);
	dim3 grid(blocks, blocks);

	//dev_erode(dev_src, dev_dst, width, height, radio, threadsPerBlock, grid);
	swapBuffers(&dev_src, &dev_dst);

}*/


// ERODE - Optimo AUTO 
void dev_erode_automaticoOnce(int radio, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, 0, 0, 0, 0, true);
	
	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);
	
	dev_erode(dev_src2, dev_dst2, width, height, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

}
/*void dev_erode_automaticoMulti(byte* dev_src, byte* dev_dst, int width, int height, int radio) {

	int threads = 32;

	dim3 threadsPerBlock(threads, threads);
	dim3 grid(width / threadsPerBlock.x, height / threadsPerBlock.y);

	//dev_erode(dev_src, dev_dst, width, height, radio, threadsPerBlock, grid);
	swapBuffers(&dev_src, &dev_dst);

}*/


// ERODE + Optimo MANUAL 
void dev_erode_twoSteps_manualOnce(int radio, int threads, int blocks, int* error) {

	hipError_t cudaStatus;

	hipDeviceSynchronize();

	cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -8;
		return;
	}

	setDimensionNumber_Threads_Blocks(2, threads, threads, blocks, blocks, false);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	dev_erode_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);

}


// ERODE + Optimo AUTO 
void dev_erode_twoSteps_automaticOnce(int radio, int* error) {

	hipError_t cudaStatus;

	hipDeviceSynchronize();

	cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -8;
		return;
	}
	
	setDimensionNumber_Threads_Blocks(2, 0, 0, 0, 0, true);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	dev_erode_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);

}



//****************************************************** LLAMADAS A DILATE ************************************************************//


// DILATE - Optimo MANUAL 
void dev_dilate_manualOnce(int radio, int threads, int blocks, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, threads, threads, blocks, blocks, false);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	dev_dilate(dev_src2, dev_dst2, width, height, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);
}


// DILATE - Optimo AUTO 
void dev_dilate_automaticOnce(int radio, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, 0, 0, 0, 0, true);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	dev_dilate(dev_src2, dev_dst2, width, height, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);
}


// DILATE + Optimo MANUAL 
void dev_dilate_twoSteps_manualOnce(int radio, int threads, int blocks, int* error) {

	hipError_t cudaStatus;
	
	hipDeviceSynchronize();
	
	cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -13;
		return;
	}

	setDimensionNumber_Threads_Blocks(2, threads, threads, blocks, blocks, false);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	dev_dilate_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);

}


// DILATE + Optimo AUTO 
void dev_dilate_twoSteps_automaticOnce(int radio, int* error) {

	hipError_t cudaStatus;

	hipDeviceSynchronize();

	cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -13;
		return;
	}

	setDimensionNumber_Threads_Blocks(2, 0, 0, 0, 0, true);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	dev_dilate_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);

}





//****************************************************** REVERSE THRESHOLD ************************************************************//

// REVERSE THRESHOLD MANUAL
void dev_reverseThreshold_manualOnce(byte min, byte max, int threads, int blocks, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(1, threads, 0, blocks, 0, false);
	int stride = threadsInX * blocksInX;

	dev_reverseThreshold(dev_src2, dev_dst2, min, max, threadsInX, blocksInY, stride, size_img, error);
	swapBuffers(&dev_src2, &dev_dst2);

}


// REVERSE THRESHOLD AUTO 
void dev_reverseThreshold_automaticOnce(byte min, byte max, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(1, 0, 0, 0, 0, true);
	int stride = threadsInX * blocksInX;

	dev_reverseThreshold(dev_src2, dev_dst2, min, max, threadsInX, blocksInX, stride, size_img, error);
	swapBuffers(&dev_src2, &dev_dst2);
}




//****************************************************** LLAMADAS A OPEN ***************************************************************//

// OPEN + Optimo MANUAL 
void dev_open_fast_manualOnce(int radio, int threads, int blocks, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, threads, threads, blocks, blocks, false);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	hipError_t cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -17;
		return;
	}
	
	dev_erode_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);
	if (error != 0)
		return;

	dev_dilate_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);
}


// OPEN + Optimo AUTO 
void dev_open_fast_automaticOnce(int radio, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, 0, 0, 0, 0, true);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	hipError_t cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -17;
		return;
	}

	dev_erode_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);
	if (error != 0)
		return;

	dev_dilate_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);

}




//********************************************************** LLAMADAS A CLOSE **************************************************************//

// CLOSE + Optimo MANUAL 
void dev_close_fast_manualOnce(int radio, int threads, int blocks, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, threads, threads, blocks, blocks, false);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	hipError_t cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -16;
		return;
	}

	dev_dilate_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);
	if (error != 0)
		return;

	dev_erode_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);

}


// CLOSE + Optimo AUTO 
void dev_close_fast_automaticOnce(int radio, int* error) {

	hipDeviceSynchronize();
	
	setDimensionNumber_Threads_Blocks(2, 0, 0, 0, 0, true);

	dim3 threadsPerBlock(threadsInX, threadsInY);
	dim3 grid(blocksInX, blocksInY);

	hipError_t cudaStatus = hipMalloc(&dev_aux2, size_img);
	if (cudaStatus != hipSuccess) {
		*error = -16;
		return;
	}

	dev_dilate_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);
	if (error != 0)
		return;

	dev_erode_twoSteps(dev_src2, dev_dst2, dev_aux2, radio, threadsPerBlock, grid, error);
	swapBuffers(&dev_src2, &dev_dst2);

	hipFree(dev_aux2);

}





//************************************************************ AUTOMASK **************************************************************//

void automask(byte *src, byte* dst, byte * dev_src, byte * dev_dst, int size, byte min, byte max, int *error) {

	hipError_t cudaSTATUS;

	*error = 1000;

	if (dev_src == NULL) {

		cudaSTATUS = hipMalloc(&dev_src, size);
		if (cudaSTATUS != hipSuccess) {
			*error = -1;
			dev_src = NULL;
			dev_dst = NULL;
		}

		cudaSTATUS = hipMalloc(&dev_dst, size);
		if (cudaSTATUS != hipSuccess) {
			*error = -2;
			dev_src = NULL;
			dev_dst = NULL;
			hipFree(dev_src);
		}
	}
	

	cudaSTATUS = hipMemcpy(dev_src, src, size, hipMemcpyHostToDevice);
	if (cudaSTATUS != hipSuccess) {
		*error = -3;
		goto Error;
	}

	int threads = 1024;
	int blocks = 640;  ///////// 65535 / 1024 = 64 * 10 
	int stride = threads * blocks;

	dev_threshold(dev_src, dev_dst, min, max, threads, blocks, stride, size, error);
	swapBuffers(&dev_src, &dev_dst);

	cudaSTATUS = hipMemcpy(dst,dev_src, size, hipMemcpyDeviceToHost);
	if (cudaSTATUS != hipSuccess) {
		*error = -4;
		goto Error;
	}

	//dst = dev_src;
	dev_src = NULL;
	hipFree(dev_src);

	dev_dst = NULL;
	hipFree(dev_dst);
	
Error:
	dev_src = NULL;
	hipFree(dev_src);

	dev_dst = NULL;
	hipFree(dev_dst);

}




//************************************************************ ERRORES **************************************************************//

/*
else if (hipError_t == hipErrorInvalidDevicePointer)
		*error = -3;
	else if (hipError_t == hipErrorInvalidMemcpyDirection)
		*error = -4;
	else if (hipError_t == hipErrorInvalidValue)
		*error = -5;

else if (cudaSTATUS == hipErrorUnsupportedLimit) {
		*error = -21;
		goto Error;
	}
else if (cudaSTATUS == cudaErrorDuplicateVariableName) {
		*error = -22;
		goto Error;
	}
	else if (cudaSTATUS == cudaErrorDuplicateTextureName) {
		*error = -23;
		goto Error;
	}
	else if (cudaSTATUS == cudaErrorDuplicateSurfaceName) {
		*error = -24;
		goto Error;
	}
	else if (cudaSTATUS == cudaErrorDevicesUnavailable) {
		*error = -25;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorInvalidImage) {
		*error = -26;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorNoBinaryForGpu) {
		*error = -27;
		goto Error;
	}
	else if (cudaSTATUS == cudaErrorIncompatibleDriverContext) {
		*error = -28;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorPeerAccessAlreadyEnabled) {
		*error = -29;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorPeerAccessNotEnabled) {
		*error = -30;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorContextAlreadyInUse) {
		*error = -31;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorProfilerDisabled) {
		*error = -32;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorProfilerNotInitialized) {
		*error = -33;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorProfilerAlreadyStarted) {
		*error = -34;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorProfilerAlreadyStopped) {
		*error = -35;
		goto Error;
	}
	else if (cudaSTATUS == cudaErrorStartupFailure) {
		*error = -36;
		goto Error;
	}
	else if (cudaSTATUS == cudaErrorApiFailureBase) {
		*error = -37;
		goto Error;
	}
	else if (cudaSTATUS == cudaErrorInvalidSurface) {
		*error = -38;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorNoDevice) {
		*error = -39;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorECCNotCorrectable) {
		*error = -40;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorSharedObjectSymbolNotFound) {
		*error = -41;
		goto Error;
	}
	else if (cudaSTATUS == hipErrorSharedObjectInitFailed) {
		*error = -42;
		goto Error;
	}
	else
	{
		*error = -1000001;
		goto Error;
	}
*/

